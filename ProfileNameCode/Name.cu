
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void shiftName(char *name)
{
    printf("Cuda:   ");
    for(int i = 8; i < 13; i++) {
        printf("%c", name[i]);
    }
    for(int i = 0; i < 4; i++) {
        printf("%c", name[i]);
    }
    printf("%c\n", name[13]);
}

void get_github_username()
{
    char *name = (char *) malloc(14*sizeof(char));
    name = (char *) "matthew peters";

    char *gpuname;
    hipMalloc((void**) &gpuname, 14*sizeof(char));
    hipMemcpy(gpuname, name, 14*sizeof(char), hipMemcpyHostToDevice);
    shiftName<<<1,1>>>(gpuname);

    hipFree(gpuname);
};

int main(int argc, char** argv)
{
    // string github_username = get_github_username();
    // cout << "Cuda:   " << github_username << endl;
    get_github_username();
    return 0;
};
